// export PATH=/usr/local/cuda/bin:$PATH
//Question : Add 2 vectors
#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void addVectors(int *gar1, int *gar2, int *gar3, int N){               
        int id = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
        if (id < N) {
            gar3[id] = gar1[id] + gar2[id]; 
        }
    

        
}
int main(){

        int N; //size of the vector
        printf("Enter the size of the vector: ");
        scanf("%d", &N);
        int *gar1, *gar2, *gar3 ;
        int *car1 = (int*)malloc(N * sizeof(int)); 
        int *car2 = (int*)malloc(N * sizeof(int)); 
        int *car3 = (int*)malloc(N * sizeof(int)); 

        for (int i = 0; i < N; ++i) {
            car1[i] = i;
            car2[i] = N - i;
        }    
        hipMalloc (&gar1, N*sizeof(int));
        hipMalloc (&gar2, N*sizeof(int));
        hipMalloc (&gar3, N*sizeof(int));
        hipMemcpy(gar1, car1, N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(gar2, car2, N * sizeof(int), hipMemcpyHostToDevice);
        //cudaMalloc (&gar2, N*sizeof(int));
        addVectors<<<1,N>>>(gar1, gar2, gar3, N);
        hipMemcpy (car3, gar3, N*sizeof(int), hipMemcpyDeviceToHost);
        for(int i=0; i<N; ++i){
                printf("%d\n", car3[i]);
        
        }
        hipFree(gar1);
        hipFree(gar2);
        hipFree(gar3);
        free(car1);
        free(car2);
        free(car3);
        return 0;

}