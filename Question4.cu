#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void square(int *matrix, int *result, int matrixSize) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int row = id / matrixSize;
    int col = id % matrixSize;

    if (row < matrixSize && col < matrixSize) {
        int sum = 0;
        for (int k = 0; k < matrixSize; ++k) {
            sum += matrix[row * matrixSize + k] * matrix[k * matrixSize + col];
        }
        result[row * matrixSize + col] = sum;
    }
}

int main() {
     int matrixSize;
    printf("The size of the matrix:");
    scanf("%d", &matrixSize);
    const int N = matrixSize * matrixSize;

    int cmat[matrixSize][matrixSize];
    int cres[matrixSize][matrixSize];

    for(int i=0;i<matrixSize;i++)
        {
            for(int j=0;j<matrixSize;j++)
                {
                    scanf("%d",&cmat[i][j]);
                }
        }

    int *gmat, *gres;
    hipMalloc(&gmat, N * sizeof(int));
    hipMalloc(&gres, N * sizeof(int));

    hipMemcpy(gmat, cmat, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    square<<<blocksPerGrid, threadsPerBlock>>>(gmat, gres, matrixSize);

    hipMemcpy(cres, gres, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < matrixSize; i++) {
        for (int j = 0; j < matrixSize; j++) {
            printf("%d ", cres[i][j]);
        }
        printf("\n");
    }

    
    hipFree(gmat);
    hipFree(gres);
    return 0;
}
