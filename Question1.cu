// export PATH=/usr/local/cuda/bin:$PATH
#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void scalar(int *gar, int N){               
        int id = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
        if (id < N) {
            gar[id] = id; 
        }
    

        
}
int main(){

        int N; //size of the vector
        printf("Enter the size of the vector: ");
        scanf("%d", &N);
        int car[N], *gar;
        hipMalloc (&gar, N*sizeof(int));
        scalar<<<1,N>>>(gar, N);
        hipMemcpy (car, gar, N*sizeof(int), hipMemcpyDeviceToHost);
        for(int i=0; i<N; ++i){
                printf("%d\n", car[i]);
        
        }
        hipFree(gar);
        free(car);
        return 0;

}
