// Add 2 2D arrays

// export PATH=/usr/local/cuda/bin:$PATH
//Question : Add 2 vectors
#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void addArrays(int *gar1, int *gar2, int *gar3, int rows, int cols){               
        int id = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
        int N = rows * cols;
        if (id < N) {
            gar3[id] = gar1[id] + gar2[id]; 
        }
    

        
}
int main(){

    int rows, cols;

    // Input the size of the 2D arrays
    printf("Enter the number of rows: ");
    scanf("%d", &rows);
    printf("Enter the number of columns: ");
    scanf("%d", &cols);

    int N = rows * cols; // Total number of elements
        int *gar1, *gar2, *gar3 ;
        int *car1 = (int*)malloc(N * sizeof(int)); 
        int *car2 = (int*)malloc(N * sizeof(int)); 
        int *car3 = (int*)malloc(N * sizeof(int)); 

        for (int i = 0; i < N; ++i) {
            car1[i] = i;
            car2[i] = N - i;
        }    
        hipMalloc (&gar1, N*sizeof(int));
        hipMalloc (&gar2, N*sizeof(int));
        hipMalloc (&gar3, N*sizeof(int));
        hipMemcpy(gar1, car1, N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(gar2, car2, N * sizeof(int), hipMemcpyHostToDevice);
        //cudaMalloc (&gar2, N*sizeof(int));
        addArrays<<<1,N>>>(gar1, gar2, gar3, rows,cols);
        hipMemcpy (car3, gar3, N*sizeof(int), hipMemcpyDeviceToHost);
        for (int i = 0; i < rows; ++i) {
            for (int j = 0; j < cols; ++j) {
                printf("%d ", car3[i * cols + j]);
            }
        }
        // Free  memory
    hipFree(gar1);
    hipFree(gar2);
    hipFree(gar3);
    free(car1);
    free(car2);
    free(car3);
        return 0;

}