#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matmul(int *gm1, int *gm2,int *gres, int r1, int c1, int r2, int c2) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int row = id / r1;
    int col = id % c2;

    if (row < r1 && col < c2) {
        int sum = 0;
        for (int k = 0; k < c1; k++) { // c1 is the same as r2
            sum += gm1[row * c1 + k] * gm2[k * c2 + col];
        }
        gres[row * c2 + col] = sum;
    }
}


int main() {
    int r1,c1,r2,c2;
    printf("Number of rows of first matrix:");
    scanf("%d",&r1);
    printf("Number of columns of first matrix:");
    scanf("%d",&c1);
    printf("Number of rows of second matrix:");
    scanf("%d",&r2);
    printf("Number of columns of second matrix:");
    scanf("%d",&c2);
    int mat_size1 = r1*c1;
    int mat_size2 = r2*c2;
    int res_size = r1 * c2;
    int cm1[r1][c1];
    int cm2[r2][c2];
    int cres[r1][c2];
    int *gm1, *gm2, *gres; 
    for(int i=0;i<r1;i++)
    {
        for(int j=0;j<c1;j++)
            {
                scanf("%d",&cm1[i][j]);
            }
    }
        for(int i=0;i<r2;i++)
        {
            for(int j=0;j<c2;j++)
                {
                    scanf("%d",&cm2[i][j]);
                }
        }
        hipMalloc(&gm1, r1*c1 * sizeof(int));
        hipMalloc(&gm2, r2*c2 * sizeof(int));
        hipMalloc(&gres, r1*c2* sizeof(int));
        hipMemcpy(gm1,cm1,r1*c1*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(gm2,cm2,r2*c2*sizeof(int),hipMemcpyHostToDevice);
        int threadsPerBlock = 256;
        int blocksPerGrid = (res_size + threadsPerBlock - 1) / threadsPerBlock;

        matmul<<<blocksPerGrid, threadsPerBlock>>>(gm1, gm2, gres, r1, c1, r2, c2 );
        hipMemcpy(cres, gres, r1*c2* sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<r1;i++)
    {
        for(int j=0;j<c2;j++)
        {
              printf("%d\t",cres[i][j]);
        }
        printf("\n");
    }
    hipFree(gm1);
    hipFree(gm2);
    hipFree(gres);
    


    return 0;
}
